#include <ctime>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define NUM_THREADS 256

int init_cuda() {
	int count;
	hipGetDeviceCount(&count);
	if (count == 0) {
		fprintf(stderr, "There is no device\n");
		return -1;
	}
	printf("There are %d device.\n", count);
	int i;
	for (i = 0; i < count; i++) {
		struct hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) break;
		}
	}
	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return -1;
	}
	hipSetDevice(i);
	return 0;
}


void matgen(float *a, int lda, int n) {
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			a[i*lda + j] = (float)rand()/RAND_MAX;
		}
	}
}


clock_t matmult(const float *a, int lda, const float *b, int ldb, 
		float *c, int ldc, int n) {
    clock_t start = clock();
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			double t = 0;
			for (int k = 0; k < n; k++) {
				t += a[i*lda + k]*b[k*ldb + j];
			}
			c[i*ldc + j] = t;
		}
	}
    clock_t end = clock();
    return end - start;
}

void compare_mat(const float *a, int lda, const float *b, int ldb, int n) {
	float max_err = 0;
	float ave_err = 0;
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			if (b[i*ldb + j] != 0) {
				float err = fabs((a[i*lda + j] - b[i*ldb + j])/b[i*ldb + j]);
				if (max_err < err) max_err = err;
				ave_err += err;
			}
		}
	}
	ave_err /= n*n;
	printf("max error: %g, average error: %g\n", max_err, ave_err);
}



__global__ static void matmultCUDA(const float *a, size_t lda, const float *b, size_t ldb,
		float *c, size_t ldc, int n) {
	extern __shared__ float data[];
	const int tid = threadIdx.x;
	const int row = blockIdx.x;

	for (int i = tid; i < n; i += blockDim.x) {
		data[i] = a[row *lda + i];
	}

    __syncthreads();

	for(int j = tid; j < n; j += blockDim.x) {
		float s = 0;
        float cc = 0;
		for (int i = 0; i < n; i++) {
            float y = data[i]*b[i*ldb + j] - cc;
			float t = s + y;
            cc = (t - s) - y;
            s = t;
		}
		c[row*ldc + j] = s;
	}
}


clock_t matMultCUDA(const float *a, int lda,
		const float *b, int ldb, float *c, int ldc, int n) {
	float *ac, *bc, *cc;
	size_t pitch_a, pitch_b, pitch_c;
	clock_t start = clock();
	hipMallocPitch((void **)&ac, &pitch_a, sizeof(float)*n, n);
	hipMallocPitch((void **)&bc, &pitch_b, sizeof(float)*n, n);
	hipMallocPitch((void **)&cc, &pitch_c, sizeof(float)*n, n);

	hipMemcpy2D(ac, pitch_a, a, sizeof(float)*lda,
			sizeof(float)*n, n, hipMemcpyHostToDevice);

	hipMemcpy2D(bc, pitch_b, b, sizeof(float)*ldb,
			sizeof(float)*n, n, hipMemcpyHostToDevice);

	int blocks = n;
	matmultCUDA<<<blocks, NUM_THREADS, sizeof(float)*n>>>
		(ac, pitch_a/sizeof(float), bc, pitch_b/sizeof(float), cc, pitch_c/sizeof(float), n);

	hipMemcpy2D(c, sizeof(float)*ldc, cc, pitch_c,
			sizeof(float)*n,n,hipMemcpyDeviceToHost);

	hipFree(ac);
	hipFree(bc);
	hipFree(cc);

	clock_t end = clock();
	return end - start;

}


int main() {
	if (init_cuda() == 0) {
		printf("CUDA initialized.\n");
	}
	else {
		printf("initialized CUDA fail!\n");
		return -1;
	}
	float *a, *b, *c, *d;
	int n = 1000;
	a = (float *)malloc(sizeof(float)*n*n);
	b = (float *)malloc(sizeof(float)*n*n);
	c = (float *)malloc(sizeof(float)*n*n);
	d = (float *)malloc(sizeof(float)*n*n);

	matgen(a, n, n);
	matgen(b, n, n);

	clock_t gpu_time = matMultCUDA(a, n, b, n, c, n, n);

	double sec = (double)gpu_time/CLOCKS_PER_SEC;
	printf("(GPU)Time used: %.2f sec(%.2lf GFLOPS)\n", sec,
			2.0*n*n*n/(sec*1E9));

	clock_t cpu_time = matmult(a,n,b,n,d,n,n);
    sec = (double)cpu_time/CLOCKS_PER_SEC;
	printf("(CPU)Time used: %.2f sec(%.2lf GFLOPS)\n", sec,
			2.0*n*n*n/(sec*1E9));
	compare_mat(c, n, d, n, n);
	return 0;
}
